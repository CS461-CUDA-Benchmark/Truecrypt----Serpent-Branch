// serpent.cpp - written and placed in the public domain by Wei Dai

/* Adapted for TrueCrypt */

#ifdef TC_WINDOWS_BOOT
#pragma optimize ("t", on)
#endif

// GWAT
//#include "Serpent.h"
#ifndef HEADER_Crypto_Serpent
#define HEADER_Crypto_Serpent

//#include "/home/arthur/Desktop/truecrypt-7.1a-source/Common/Tcdefs.h"
/*
 Legal Notice: Some portions of the source code contained in this file were
 derived from the source code of Encryption for the Masses 2.02a, which is
 Copyright (c) 1998-2000 Paul Le Roux and which is governed by the 'License
 Agreement for Encryption for the Masses'. Modifications and additions to
 the original source code (contained in this file) and all other portions
 of this file are Copyright (c) 2003-2010 TrueCrypt Developers Association
 and are governed by the TrueCrypt License 3.0 the full text of which is
 contained in the file License.txt included in TrueCrypt binary and source
 code distribution packages. */

#ifndef TCDEFS_H
#define TCDEFS_H

#define TC_APP_NAME						"TrueCrypt"

// Version displayed to user 
#define VERSION_STRING					"7.1a"

// Version number to compare against driver
#define VERSION_NUM						0x071a

// Release date
#define TC_STR_RELEASE_DATE				"February 7, 2012"
#define TC_RELEASE_DATE_YEAR			2012
#define TC_RELEASE_DATE_MONTH			2

#define BYTES_PER_KB                    1024LL
#define BYTES_PER_MB                    1048576LL
#define BYTES_PER_GB                    1073741824LL
#define BYTES_PER_TB                    1099511627776LL
#define BYTES_PER_PB                    1125899906842624LL

/* GUI/driver errors */

#define WIDE(x) (LPWSTR)L##x

#ifdef _MSC_VER

typedef __int8 int8;
typedef __int16 int16;
typedef __int32 int32;
typedef unsigned __int8 byte;
typedef unsigned __int16 uint16;
typedef unsigned __int32 uint32;

#ifdef TC_NO_COMPILER_INT64
typedef unsigned __int32	TC_LARGEST_COMPILER_UINT;
#else
typedef unsigned __int64	TC_LARGEST_COMPILER_UINT;
typedef __int64 int64;
typedef unsigned __int64 uint64;
#endif

#else // !_MSC_VER

#include <inttypes.h>
#include <limits.h>

typedef int8_t int8;
typedef int16_t int16;
typedef int32_t int32;
typedef int64_t int64;
typedef uint8_t byte;
typedef uint16_t uint16;
typedef uint32_t uint32;
typedef uint64_t uint64;

#if UCHAR_MAX != 0xffU
#error UCHAR_MAX != 0xff
#endif
#define __int8 char

#if USHRT_MAX != 0xffffU
#error USHRT_MAX != 0xffff
#endif
#define __int16 short

#if UINT_MAX != 0xffffffffU
#error UINT_MAX != 0xffffffff
#endif
#define __int32 int

typedef uint64 TC_LARGEST_COMPILER_UINT;

#define BOOL int
#ifndef FALSE
#define FALSE 0
#define TRUE 1
#endif

#endif // !_MSC_VER

#define TC_INT_TYPES_DEFINED

// Integer types required by Cryptolib
typedef unsigned __int8 uint_8t;
typedef unsigned __int16 uint_16t;
typedef unsigned __int32 uint_32t;
#ifndef TC_NO_COMPILER_INT64
typedef uint64 uint_64t;
#endif

typedef union 
{
	struct 
	{
		unsigned __int32 LowPart;
		unsigned __int32 HighPart;
	};
#ifndef TC_NO_COMPILER_INT64
	uint64 Value;
#endif

} UINT64_STRUCT;

#ifdef TC_WINDOWS_BOOT

#	ifdef  __cplusplus
extern "C"
#	endif
void ThrowFatalException (int line);

#	define TC_THROW_FATAL_EXCEPTION	ThrowFatalException (__LINE__)
#elif defined (TC_WINDOWS_DRIVER)
#	define TC_THROW_FATAL_EXCEPTION KeBugCheckEx (SECURITY_SYSTEM, __LINE__, 0, 0, 'TC')
#else
#	define TC_THROW_FATAL_EXCEPTION	*(char *) 0 = 0
#endif

#ifdef TC_WINDOWS_DRIVER

#include <ntifs.h>
#include <ntddk.h>		/* Standard header file for nt drivers */
#include <ntdddisk.h>		/* Standard I/O control codes  */

#define TCalloc(size) ((void *) ExAllocatePoolWithTag( NonPagedPool, size, 'MMCT' ))
#define TCfree(memblock) ExFreePoolWithTag( memblock, 'MMCT' )

#define DEVICE_DRIVER

#ifndef BOOL
typedef int BOOL;
#endif

#ifndef TRUE
#define TRUE 1
#endif

#ifndef FALSE
#define FALSE !TRUE
#endif

#else				/* !TC_WINDOWS_DRIVER */

#define TCalloc malloc
#define TCfree free

#ifdef _WIN32

#ifndef TC_LOCAL_WIN32_WINNT_OVERRIDE
#	undef _WIN32_WINNT
#	define	_WIN32_WINNT 0x0501	/* Does not apply to the driver */
#endif

#include <windows.h>		/* Windows header */
#include <commctrl.h>		/* The common controls */
#include <process.h>		/* Process control */
#include <winioctl.h>
#include <stdio.h>		/* For sprintf */

#endif				/* _WIN32 */

#endif				/* !TC_WINDOWS_DRIVER */

#ifndef TC_TO_STRING
#	define TC_TO_STRING2(n) #n
#	define TC_TO_STRING(n) TC_TO_STRING2(n)
#endif

#ifdef DEVICE_DRIVER
#	if defined (DEBUG) || 0
#		if 1 // DbgPrintEx is not available on Windows 2000
#			define Dump DbgPrint
#		else
#			define Dump(...) DbgPrintEx (DPFLTR_IHVDRIVER_ID, DPFLTR_ERROR_LEVEL, __VA_ARGS__)
#		endif
#		define DumpMem(...) DumpMemory (__VA_ARGS__)
#	else
#		define Dump(...)
#		define DumpMem(...)
#	endif
#endif

#if !defined (trace_msg) && !defined (TC_WINDOWS_BOOT)
#	ifdef DEBUG
#		ifdef DEVICE_DRIVER
#			define trace_msg Dump
#		elif defined (_WIN32)
#			define trace_msg(...) do { char msg[2048]; _snprintf (msg, sizeof (msg), __VA_ARGS__); OutputDebugString (msg); } while (0)
#		endif
#		define trace_point trace_msg (__FUNCTION__ ":" TC_TO_STRING(__LINE__) "\n")
#	else
#		define trace_msg(...)
#		define trace_point
#	endif
#endif

#ifdef DEVICE_DRIVER
#	define TC_EVENT KEVENT
#	define TC_WAIT_EVENT(EVENT) KeWaitForSingleObject (&EVENT, Executive, KernelMode, FALSE, NULL)
#elif defined (_WIN32)
#	define TC_EVENT HANDLE
#	define TC_WAIT_EVENT(EVENT) WaitForSingleObject (EVENT, INFINITE)
#endif

#ifdef _WIN32
#define burn(mem,size) do { volatile char *burnm = (volatile char *)(mem); int burnc = size; RtlSecureZeroMemory (mem, size); while (burnc--) *burnm++ = 0; } while (0)
#else
#define burn(mem,size) do { volatile char *burnm = (volatile char *)(mem); int burnc = size; while (burnc--) *burnm++ = 0; } while (0)
#endif

// The size of the memory area to wipe is in bytes amd it must be a multiple of 8.
#ifndef TC_NO_COMPILER_INT64
#	define FAST_ERASE64(mem,size) do { volatile uint64 *burnm = (volatile uint64 *)(mem); int burnc = size >> 3; while (burnc--) *burnm++ = 0; } while (0)
#else
#	define FAST_ERASE64(mem,size) do { volatile unsigned __int32 *burnm = (volatile unsigned __int32 *)(mem); int burnc = size >> 2; while (burnc--) *burnm++ = 0; } while (0)
#endif

#ifdef TC_WINDOWS_BOOT
#	ifndef max
#		define max(a,b) (((a) > (b)) ? (a) : (b))
#	endif

#	ifdef  __cplusplus
extern "C"
#	endif
void EraseMemory (void *memory, int size);

#	undef burn
#	define burn EraseMemory
#endif

#ifdef MAX_PATH
#define TC_MAX_PATH		MAX_PATH
#else
#define TC_MAX_PATH		260	/* Includes the null terminator */
#endif

#define TC_STR_RELEASED_BY "Released by TrueCrypt Foundation on " TC_STR_RELEASE_DATE

#define MAX_URL_LENGTH	2084 /* Internet Explorer limit. Includes the terminating null character. */

#define TC_HOMEPAGE "http://www.truecrypt.org/"
#define TC_APPLINK "http://www.truecrypt.org/applink?version=" VERSION_STRING
#define TC_APPLINK_SECURE "https://www.truecrypt.org/applink?version=" VERSION_STRING

enum
{
	/* WARNING: ADD ANY NEW CODES AT THE END (DO NOT INSERT THEM BETWEEN EXISTING). DO *NOT* DELETE ANY 
	EXISTING CODES! Changing these values or their meanings may cause incompatibility with other versions
	(for example, if a new version of the TrueCrypt installer receives an error code from an installed 
	driver whose version is lower, it will report and interpret the error incorrectly). */

	ERR_SUCCESS								= 0,
	ERR_OS_ERROR							= 1,
	ERR_OUTOFMEMORY							= 2,
	ERR_PASSWORD_WRONG						= 3,
	ERR_VOL_FORMAT_BAD						= 4,
	ERR_DRIVE_NOT_FOUND						= 5,
	ERR_FILES_OPEN							= 6,
	ERR_VOL_SIZE_WRONG						= 7,
	ERR_COMPRESSION_NOT_SUPPORTED			= 8,
	ERR_PASSWORD_CHANGE_VOL_TYPE			= 9,
	ERR_PASSWORD_CHANGE_VOL_VERSION			= 10,
	ERR_VOL_SEEKING							= 11,
	ERR_VOL_WRITING							= 12,
	ERR_FILES_OPEN_LOCK						= 13,
	ERR_VOL_READING							= 14,
	ERR_DRIVER_VERSION						= 15,
	ERR_NEW_VERSION_REQUIRED				= 16,
	ERR_CIPHER_INIT_FAILURE					= 17,
	ERR_CIPHER_INIT_WEAK_KEY				= 18,
	ERR_SELF_TESTS_FAILED					= 19,
	ERR_SECTOR_SIZE_INCOMPATIBLE			= 20,
	ERR_VOL_ALREADY_MOUNTED					= 21,
	ERR_NO_FREE_DRIVES						= 22,
	ERR_FILE_OPEN_FAILED					= 23,
	ERR_VOL_MOUNT_FAILED					= 24,
	DEPRECATED_ERR_INVALID_DEVICE			= 25,
	ERR_ACCESS_DENIED						= 26,
	ERR_MODE_INIT_FAILED					= 27,
	ERR_DONT_REPORT							= 28,
	ERR_ENCRYPTION_NOT_COMPLETED			= 29,
	ERR_PARAMETER_INCORRECT					= 30,
	ERR_SYS_HIDVOL_HEAD_REENC_MODE_WRONG	= 31,
	ERR_NONSYS_INPLACE_ENC_INCOMPLETE		= 32,
	ERR_USER_ABORT							= 33
};

#endif 	// #ifndef TCDEFS_H

#ifdef __cplusplus
extern "C"
{
#endif

void serpent_set_key(const unsigned __int8 userKey[], int keylen, unsigned __int8 *ks);
void serpent_encrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks);
void serpent_decrypt(const unsigned __int8 *inBlock,  unsigned __int8 *outBlock, unsigned __int8 *ks);

#ifdef __cplusplus
}
#endif

#endif // HEADER_Crypto_Serpent
//END GWAT

// GWAT
//#include "/home/arthur/Desktop/truecrypt-7.1a-source/Common/Endian.h"
/*
 Legal Notice: Some portions of the source code contained in this file were
 derived from the source code of Encryption for the Masses 2.02a, which is
 Copyright (c) 1998-2000 Paul Le Roux and which is governed by the 'License
 Agreement for Encryption for the Masses'. Modifications and additions to
 the original source code (contained in this file) and all other portions
 of this file are Copyright (c) 2003-2009 TrueCrypt Developers Association
 and are governed by the TrueCrypt License 3.0 the full text of which is
 contained in the file License.txt included in TrueCrypt binary and source
 code distribution packages. */

#ifndef TC_ENDIAN_H
#define TC_ENDIAN_H

//#include "/home/arthur/Desktop/truecrypt-7.1a-source/Common/Tcdefs.h"

#if defined(__cplusplus)
extern "C"
{
#endif

#ifdef _WIN32

#	ifndef LITTLE_ENDIAN
#		define LITTLE_ENDIAN 1234
#	endif
#	ifndef BYTE_ORDER
#		define BYTE_ORDER LITTLE_ENDIAN
#	endif

#elif !defined(BYTE_ORDER)

#	ifdef TC_MACOSX
#		include <machine/endian.h>
#	elif defined (TC_BSD)
#		include <sys/endian.h>
#	elif defined (TC_SOLARIS)
#		include <sys/types.h>
#		define LITTLE_ENDIAN 1234
#		define BIG_ENDIAN 4321
#		ifdef _BIG_ENDIAN
#			define BYTE_ORDER BIG_ENDIAN
#		else
#			define BYTE_ORDER LITTLE_ENDIAN
#		endif
#	else
#		include <endian.h>
#	endif

#	ifndef BYTE_ORDER
#		ifndef __BYTE_ORDER
#			error Byte order cannot be determined (BYTE_ORDER undefined)
#		endif

#		define BYTE_ORDER __BYTE_ORDER
#	endif

#	ifndef LITTLE_ENDIAN
#		define LITTLE_ENDIAN __LITTLE_ENDIAN
#	endif

#	ifndef BIG_ENDIAN
#		define BIG_ENDIAN __BIG_ENDIAN
#	endif

#endif // !BYTE_ORDER

/* Macros to read and write 16, 32, and 64-bit quantities in a portable manner.
   These functions are implemented as macros rather than true functions as
   the need to adjust the memory pointers makes them somewhat painful to call
   in user code */

#define mputInt64(memPtr,data) \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 56 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 48 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 40 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 32 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 24 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 16 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 8 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( data ) & 0xFF )

#define mputLong(memPtr,data) \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 24 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 16 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 8 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( data ) & 0xFF )

#define mputWord(memPtr,data) \
	*memPtr++ = ( unsigned char ) ( ( ( data ) >> 8 ) & 0xFF ), \
	*memPtr++ = ( unsigned char ) ( ( data ) & 0xFF )

#define mputByte(memPtr,data)	\
	*memPtr++ = ( unsigned char ) data

#define mputBytes(memPtr,data,len)  \
	memcpy (memPtr,data,len); \
	memPtr += len;

#define mgetInt64(memPtr) 		\
	( memPtr += 8, ( ( unsigned __int64 ) memPtr[ -8 ] << 56 ) | ( ( unsigned __int64 ) memPtr[ -7 ] << 48 ) | \
	( ( unsigned __int64 ) memPtr[ -6 ] << 40 ) | ( ( unsigned __int64 ) memPtr[ -5 ] << 32 ) | \
	( ( unsigned __int64 ) memPtr[ -4 ] << 24 ) | ( ( unsigned __int64 ) memPtr[ -3 ] << 16 ) | \
	  ( ( unsigned __int64 ) memPtr[ -2 ] << 8 ) | ( unsigned __int64 ) memPtr[ -1 ] )

#define mgetLong(memPtr) 		\
	( memPtr += 4, ( ( unsigned __int32 ) memPtr[ -4 ] << 24 ) | ( ( unsigned __int32 ) memPtr[ -3 ] << 16 ) | \
	  ( ( unsigned __int32 ) memPtr[ -2 ] << 8 ) | ( unsigned __int32 ) memPtr[ -1 ] )

#define mgetWord(memPtr) 		\
	( memPtr += 2, ( unsigned short ) memPtr[ -2 ] << 8 ) | ( ( unsigned short ) memPtr[ -1 ] ) 

#define mgetByte(memPtr)		\
	( ( unsigned char ) *memPtr++ )

#if BYTE_ORDER == BIG_ENDIAN
#	define LE16(x) MirrorBytes16(x)
#	define LE32(x) MirrorBytes32(x)
#	define LE64(x) MirrorBytes64(x)
#else
#	define LE16(x) (x)
#	define LE32(x) (x)
#	define LE64(x) (x)
#endif

#if BYTE_ORDER == LITTLE_ENDIAN
#	define BE16(x) MirrorBytes16(x)
#	define BE32(x) MirrorBytes32(x)
#	define BE64(x) MirrorBytes64(x)
#else
#	define BE16(x) (x)
#	define BE32(x) (x)
#	define BE64(x) (x)
#endif

unsigned __int16 MirrorBytes16 (unsigned __int16 x);
unsigned __int32 MirrorBytes32 (unsigned __int32 x);
#ifndef TC_NO_COMPILER_INT64
uint64 MirrorBytes64 (uint64 x);
#endif 
void LongReverse ( unsigned __int32 *buffer , unsigned byteCount );

#if defined(__cplusplus)
}
#endif

#endif /* TC_ENDIAN_H */
//END GWAT

#include <memory.h>

// GWAT
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#if defined(_WIN32) && !defined(_DEBUG)
#include <stdlib.h>
#define rotlFixed _rotl
#define rotrFixed _rotr
#else
#define rotlFixed(x,n)   (((x) << (n)) | ((x) >> (32 - (n))))
#define rotrFixed(x,n)   (((x) >> (n)) | ((x) << (32 - (n))))
#endif

// linear transformation
#define LT(i,a,b,c,d,e)	{\
	a = rotlFixed(a, 13);	\
	c = rotlFixed(c, 3); 	\
	d = rotlFixed(d ^ c ^ (a << 3), 7); 	\
	b = rotlFixed(b ^ a ^ c, 1); 	\
	a = rotlFixed(a ^ b ^ d, 5); 		\
	c = rotlFixed(c ^ d ^ (b << 7), 22);}

// inverse linear transformation
#define ILT(i,a,b,c,d,e)	{\
	c = rotrFixed(c, 22);	\
	a = rotrFixed(a, 5); 	\
	c ^= d ^ (b << 7);	\
	a ^= b ^ d; 		\
	b = rotrFixed(b, 1); 	\
	d = rotrFixed(d, 7) ^ c ^ (a << 3);	\
	b ^= a ^ c; 		\
	c = rotrFixed(c, 3); 	\
	a = rotrFixed(a, 13);}

// order of output from S-box functions
#define beforeS0(f) f(0,a,b,c,d,e)
#define afterS0(f) f(1,b,e,c,a,d)
#define afterS1(f) f(2,c,b,a,e,d)
#define afterS2(f) f(3,a,e,b,d,c)
#define afterS3(f) f(4,e,b,d,c,a)
#define afterS4(f) f(5,b,a,e,c,d)
#define afterS5(f) f(6,a,c,b,e,d)
#define afterS6(f) f(7,a,c,d,b,e)
#define afterS7(f) f(8,d,e,b,a,c)

// order of output from inverse S-box functions
#define beforeI7(f) f(8,a,b,c,d,e)
#define afterI7(f) f(7,d,a,b,e,c)
#define afterI6(f) f(6,a,b,c,e,d)
#define afterI5(f) f(5,b,d,e,c,a)
#define afterI4(f) f(4,b,c,e,a,d)
#define afterI3(f) f(3,a,b,e,c,d)
#define afterI2(f) f(2,b,d,e,c,a)
#define afterI1(f) f(1,a,b,c,e,d)
#define afterI0(f) f(0,a,d,b,e,c)

// The instruction sequences for the S-box functions 
// come from Dag Arne Osvik's paper "Speeding up Serpent".

#define S0(i, r0, r1, r2, r3, r4) \
       {           \
    r3 ^= r0;   \
    r4 = r1;   \
    r1 &= r3;   \
    r4 ^= r2;   \
    r1 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r4;   \
    r4 ^= r3;   \
    r3 ^= r2;   \
    r2 |= r1;   \
    r2 ^= r4;   \
    r4 = ~r4;      \
    r4 |= r1;   \
    r1 ^= r3;   \
    r1 ^= r4;   \
    r3 |= r0;   \
    r1 ^= r3;   \
    r4 ^= r3;   \
            }

#define I0(i, r0, r1, r2, r3, r4) \
       {           \
    r2 = ~r2;      \
    r4 = r1;   \
    r1 |= r0;   \
    r4 = ~r4;      \
    r1 ^= r2;   \
    r2 |= r4;   \
    r1 ^= r3;   \
    r0 ^= r4;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r4 ^= r0;   \
    r0 |= r1;   \
    r0 ^= r2;   \
    r3 ^= r4;   \
    r2 ^= r1;   \
    r3 ^= r0;   \
    r3 ^= r1;   \
    r2 &= r3;   \
    r4 ^= r2;   \
            }

#define S1(i, r0, r1, r2, r3, r4) \
       {           \
    r0 = ~r0;      \
    r2 = ~r2;      \
    r4 = r0;   \
    r0 &= r1;   \
    r2 ^= r0;   \
    r0 |= r3;   \
    r3 ^= r2;   \
    r1 ^= r0;   \
    r0 ^= r4;   \
    r4 |= r1;   \
    r1 ^= r3;   \
    r2 |= r0;   \
    r2 &= r4;   \
    r0 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r0;   \
    r0 &= r2;   \
    r0 ^= r4;   \
            }

#define I1(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r1;   \
    r1 ^= r3;   \
    r3 &= r1;   \
    r4 ^= r2;   \
    r3 ^= r0;   \
    r0 |= r1;   \
    r2 ^= r3;   \
    r0 ^= r4;   \
    r0 |= r2;   \
    r1 ^= r3;   \
    r0 ^= r1;   \
    r1 |= r3;   \
    r1 ^= r0;   \
    r4 = ~r4;      \
    r4 ^= r1;   \
    r1 |= r0;   \
    r1 ^= r0;   \
    r1 |= r4;   \
    r3 ^= r1;   \
            }

#define S2(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r0;   \
    r0 &= r2;   \
    r0 ^= r3;   \
    r2 ^= r1;   \
    r2 ^= r0;   \
    r3 |= r4;   \
    r3 ^= r1;   \
    r4 ^= r2;   \
    r1 = r3;   \
    r3 |= r4;   \
    r3 ^= r0;   \
    r0 &= r1;   \
    r4 ^= r0;   \
    r1 ^= r3;   \
    r1 ^= r4;   \
    r4 = ~r4;      \
            }

#define I2(i, r0, r1, r2, r3, r4) \
       {           \
    r2 ^= r3;   \
    r3 ^= r0;   \
    r4 = r3;   \
    r3 &= r2;   \
    r3 ^= r1;   \
    r1 |= r2;   \
    r1 ^= r4;   \
    r4 &= r3;   \
    r2 ^= r3;   \
    r4 &= r0;   \
    r4 ^= r2;   \
    r2 &= r1;   \
    r2 |= r0;   \
    r3 = ~r3;      \
    r2 ^= r3;   \
    r0 ^= r3;   \
    r0 &= r1;   \
    r3 ^= r4;   \
    r3 ^= r0;   \
            }

#define S3(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r0;   \
    r0 |= r3;   \
    r3 ^= r1;   \
    r1 &= r4;   \
    r4 ^= r2;   \
    r2 ^= r3;   \
    r3 &= r0;   \
    r4 |= r1;   \
    r3 ^= r4;   \
    r0 ^= r1;   \
    r4 &= r0;   \
    r1 ^= r3;   \
    r4 ^= r2;   \
    r1 |= r0;   \
    r1 ^= r2;   \
    r0 ^= r3;   \
    r2 = r1;   \
    r1 |= r3;   \
    r1 ^= r0;   \
            }

#define I3(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r0;   \
    r0 &= r4;   \
    r4 ^= r3;   \
    r3 |= r1;   \
    r3 ^= r2;   \
    r0 ^= r4;   \
    r2 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r1;   \
    r4 ^= r2;   \
    r2 &= r3;   \
    r1 |= r3;   \
    r1 ^= r2;   \
    r4 ^= r0;   \
    r2 ^= r4;   \
            }

#define S4(i, r0, r1, r2, r3, r4) \
       {           \
    r1 ^= r3;   \
    r3 = ~r3;      \
    r2 ^= r3;   \
    r3 ^= r0;   \
    r4 = r1;   \
    r1 &= r3;   \
    r1 ^= r2;   \
    r4 ^= r3;   \
    r0 ^= r4;   \
    r2 &= r4;   \
    r2 ^= r0;   \
    r0 &= r1;   \
    r3 ^= r0;   \
    r4 |= r1;   \
    r4 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r2;   \
    r2 &= r3;   \
    r0 = ~r0;      \
    r4 ^= r2;   \
            }

#define I4(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 &= r3;   \
    r2 ^= r1;   \
    r1 |= r3;   \
    r1 &= r0;   \
    r4 ^= r2;   \
    r4 ^= r1;   \
    r1 &= r2;   \
    r0 = ~r0;      \
    r3 ^= r4;   \
    r1 ^= r3;   \
    r3 &= r0;   \
    r3 ^= r2;   \
    r0 ^= r1;   \
    r2 &= r0;   \
    r3 ^= r0;   \
    r2 ^= r4;   \
    r2 |= r3;   \
    r3 ^= r0;   \
    r2 ^= r1;   \
            }

#define S5(i, r0, r1, r2, r3, r4) \
       {           \
    r0 ^= r1;   \
    r1 ^= r3;   \
    r3 = ~r3;      \
    r4 = r1;   \
    r1 &= r0;   \
    r2 ^= r3;   \
    r1 ^= r2;   \
    r2 |= r4;   \
    r4 ^= r3;   \
    r3 &= r1;   \
    r3 ^= r0;   \
    r4 ^= r1;   \
    r4 ^= r2;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r2 = ~r2;      \
    r0 ^= r4;   \
    r4 |= r3;   \
    r2 ^= r4;   \
            }

#define I5(i, r0, r1, r2, r3, r4) \
       {           \
    r1 = ~r1;      \
    r4 = r3;   \
    r2 ^= r1;   \
    r3 |= r0;   \
    r3 ^= r2;   \
    r2 |= r1;   \
    r2 &= r0;   \
    r4 ^= r3;   \
    r2 ^= r4;   \
    r4 |= r0;   \
    r4 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r3;   \
    r4 ^= r2;   \
    r3 &= r4;   \
    r4 ^= r1;   \
    r3 ^= r0;   \
    r3 ^= r4;   \
    r4 = ~r4;      \
            }

#define S6(i, r0, r1, r2, r3, r4) \
       {           \
    r2 = ~r2;      \
    r4 = r3;   \
    r3 &= r0;   \
    r0 ^= r4;   \
    r3 ^= r2;   \
    r2 |= r4;   \
    r1 ^= r3;   \
    r2 ^= r0;   \
    r0 |= r1;   \
    r2 ^= r1;   \
    r4 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r2;   \
    r4 ^= r3;   \
    r4 ^= r0;   \
    r3 = ~r3;      \
    r2 &= r4;   \
    r2 ^= r3;   \
            }

#define I6(i, r0, r1, r2, r3, r4) \
       {           \
    r0 ^= r2;   \
    r4 = r2;   \
    r2 &= r0;   \
    r4 ^= r3;   \
    r2 = ~r2;      \
    r3 ^= r1;   \
    r2 ^= r3;   \
    r4 |= r0;   \
    r0 ^= r2;   \
    r3 ^= r4;   \
    r4 ^= r1;   \
    r1 &= r3;   \
    r1 ^= r0;   \
    r0 ^= r3;   \
    r0 |= r2;   \
    r3 ^= r1;   \
    r4 ^= r0;   \
            }

#define S7(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 &= r1;   \
    r2 ^= r3;   \
    r3 &= r1;   \
    r4 ^= r2;   \
    r2 ^= r1;   \
    r1 ^= r0;   \
    r0 |= r4;   \
    r0 ^= r2;   \
    r3 ^= r1;   \
    r2 ^= r3;   \
    r3 &= r0;   \
    r3 ^= r4;   \
    r4 ^= r2;   \
    r2 &= r0;   \
    r4 = ~r4;      \
    r2 ^= r4;   \
    r4 &= r0;   \
    r1 ^= r3;   \
    r4 ^= r1;   \
            }

#define I7(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r2 = ~r2;      \
    r4 |= r3;   \
    r3 ^= r1;   \
    r1 |= r0;   \
    r0 ^= r2;   \
    r2 &= r4;   \
    r1 ^= r2;   \
    r2 ^= r0;   \
    r0 |= r2;   \
    r3 &= r4;   \
    r0 ^= r3;   \
    r4 ^= r1;   \
    r3 ^= r4;   \
    r4 |= r0;   \
    r3 ^= r2;   \
    r4 ^= r2;   \
            }

// key xor
#define KX(r, a, b, c, d, e)	{\
	a ^= k[4 * r + 0]; \
	b ^= k[4 * r + 1]; \
	c ^= k[4 * r + 2]; \
	d ^= k[4 * r + 3];}


#ifdef TC_MINIMIZE_CODE_SIZE

static void S0f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{
	*r3 ^= *r0;
	*r4 = *r1;
	*r1 &= *r3;
	*r4 ^= *r2;
	*r1 ^= *r0;
	*r0 |= *r3;
	*r0 ^= *r4;
	*r4 ^= *r3;
	*r3 ^= *r2;
	*r2 |= *r1;
	*r2 ^= *r4;
	*r4 = ~*r4;
	*r4 |= *r1;
	*r1 ^= *r3;
	*r1 ^= *r4;
	*r3 |= *r0;
	*r1 ^= *r3;
	*r4 ^= *r3;
}

static void S1f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
    *r0 = ~*r0;   
    *r2 = ~*r2;   
    *r4 = *r0;
    *r0 &= *r1;
    *r2 ^= *r0;
    *r0 |= *r3;
    *r3 ^= *r2;
    *r1 ^= *r0;
    *r0 ^= *r4;
    *r4 |= *r1;
    *r1 ^= *r3;
    *r2 |= *r0;
    *r2 &= *r4;
    *r0 ^= *r1;
    *r1 &= *r2;
    *r1 ^= *r0;
    *r0 &= *r2;
    *r0 ^= *r4;
}

static void S2f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r4 = *r0;
	*r0 &= *r2;
	*r0 ^= *r3;
	*r2 ^= *r1;
	*r2 ^= *r0;
	*r3 |= *r4;
	*r3 ^= *r1;
	*r4 ^= *r2;
	*r1 = *r3;
	*r3 |= *r4;
	*r3 ^= *r0;
	*r0 &= *r1;
	*r4 ^= *r0;
	*r1 ^= *r3;
	*r1 ^= *r4;
	*r4 = ~*r4;   
}

static void S3f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r4 = *r0;
	*r0 |= *r3;
	*r3 ^= *r1;
	*r1 &= *r4;
	*r4 ^= *r2;
	*r2 ^= *r3;
	*r3 &= *r0;
	*r4 |= *r1;
	*r3 ^= *r4;
	*r0 ^= *r1;
	*r4 &= *r0;
	*r1 ^= *r3;
	*r4 ^= *r2;
	*r1 |= *r0;
	*r1 ^= *r2;
	*r0 ^= *r3;
	*r2 = *r1;
	*r1 |= *r3;
	*r1 ^= *r0;
}

static void S4f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r1 ^= *r3;
	*r3 = ~*r3;   
	*r2 ^= *r3;
	*r3 ^= *r0;
	*r4 = *r1;
	*r1 &= *r3;
	*r1 ^= *r2;
	*r4 ^= *r3;
	*r0 ^= *r4;
	*r2 &= *r4;
	*r2 ^= *r0;
	*r0 &= *r1;
	*r3 ^= *r0;
	*r4 |= *r1;
	*r4 ^= *r0;
	*r0 |= *r3;
	*r0 ^= *r2;
	*r2 &= *r3;
	*r0 = ~*r0;   
	*r4 ^= *r2;
}

static void S5f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r0 ^= *r1;
	*r1 ^= *r3;
	*r3 = ~*r3;   
	*r4 = *r1;
	*r1 &= *r0;
	*r2 ^= *r3;
	*r1 ^= *r2;
	*r2 |= *r4;
	*r4 ^= *r3;
	*r3 &= *r1;
	*r3 ^= *r0;
	*r4 ^= *r1;
	*r4 ^= *r2;
	*r2 ^= *r0;
	*r0 &= *r3;
	*r2 = ~*r2;   
	*r0 ^= *r4;
	*r4 |= *r3;
	*r2 ^= *r4;
}

static void S6f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r2 = ~*r2;   
	*r4 = *r3;
	*r3 &= *r0;
	*r0 ^= *r4;
	*r3 ^= *r2;
	*r2 |= *r4;
	*r1 ^= *r3;
	*r2 ^= *r0;
	*r0 |= *r1;
	*r2 ^= *r1;
	*r4 ^= *r0;
	*r0 |= *r3;
	*r0 ^= *r2;
	*r4 ^= *r3;
	*r4 ^= *r0;
	*r3 = ~*r3;   
	*r2 &= *r4;
	*r2 ^= *r3;
}

static void S7f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r4 = *r2;
	*r2 &= *r1;
	*r2 ^= *r3;
	*r3 &= *r1;
	*r4 ^= *r2;
	*r2 ^= *r1;
	*r1 ^= *r0;
	*r0 |= *r4;
	*r0 ^= *r2;
	*r3 ^= *r1;
	*r2 ^= *r3;
	*r3 &= *r0;
	*r3 ^= *r4;
	*r4 ^= *r2;
	*r2 &= *r0;
	*r4 = ~*r4;   
	*r2 ^= *r4;
	*r4 &= *r0;
	*r1 ^= *r3;
	*r4 ^= *r1;
}

static void KXf (const unsigned __int32 *k, unsigned int r, unsigned __int32 *a, unsigned __int32 *b, unsigned __int32 *c, unsigned __int32 *d)
{
	*a ^= k[r];
	*b ^= k[r + 1];
	*c ^= k[r + 2];
	*d ^= k[r + 3];
}

#endif // TC_MINIMIZE_CODE_SIZE

#ifndef TC_MINIMIZE_CODE_SIZE

void serpent_set_key(const unsigned __int8 userKey[], int keylen, unsigned __int8 *ks)
{
	unsigned __int32 a,b,c,d,e;
	unsigned __int32 *k = (unsigned __int32 *)ks;
	unsigned __int32 t;
	int i;

	for (i = 0; i < keylen / (int)sizeof(__int32); i++)
		k[i] = LE32(((unsigned __int32*)userKey)[i]);

	if (keylen < 32)
		k[keylen/4] |= (unsigned __int32)1 << ((keylen%4)*8);

	k += 8;
	t = k[-1];
	for (i = 0; i < 132; ++i)
		k[i] = t = rotlFixed(k[i-8] ^ k[i-5] ^ k[i-3] ^ t ^ 0x9e3779b9 ^ i, 11);
	k -= 20;

#define LK(r, a, b, c, d, e)	{\
	a = k[(8-r)*4 + 0];		\
	b = k[(8-r)*4 + 1];		\
	c = k[(8-r)*4 + 2];		\
	d = k[(8-r)*4 + 3];}

#define SK(r, a, b, c, d, e)	{\
	k[(8-r)*4 + 4] = a;		\
	k[(8-r)*4 + 5] = b;		\
	k[(8-r)*4 + 6] = c;		\
	k[(8-r)*4 + 7] = d;}	\

	for (i=0; i<4; i++)
	{
		afterS2(LK); afterS2(S3); afterS3(SK);
		afterS1(LK); afterS1(S2); afterS2(SK);
		afterS0(LK); afterS0(S1); afterS1(SK);
		beforeS0(LK); beforeS0(S0); afterS0(SK);
		k += 8*4;
		afterS6(LK); afterS6(S7); afterS7(SK);
		afterS5(LK); afterS5(S6); afterS6(SK);
		afterS4(LK); afterS4(S5); afterS5(SK);
		afterS3(LK); afterS3(S4); afterS4(SK);
	}
	afterS2(LK); afterS2(S3); afterS3(SK);
}

#else // TC_MINIMIZE_CODE_SIZE

static void LKf (unsigned __int32 *k, unsigned int r, unsigned __int32 *a, unsigned __int32 *b, unsigned __int32 *c, unsigned __int32 *d)
{
	*a = k[r];
	*b = k[r + 1];
	*c = k[r + 2];
	*d = k[r + 3];
}

static void SKf (unsigned __int32 *k, unsigned int r, unsigned __int32 *a, unsigned __int32 *b, unsigned __int32 *c, unsigned __int32 *d)
{
	k[r + 4] = *a;
	k[r + 5] = *b;
	k[r + 6] = *c;
	k[r + 7] = *d;
}

void serpent_set_key(const unsigned __int8 userKey[], int keylen, unsigned __int8 *ks)
{
	unsigned __int32 a,b,c,d,e;
	unsigned __int32 *k = (unsigned __int32 *)ks;
	unsigned __int32 t;
	int i;

	for (i = 0; i < keylen / (int)sizeof(__int32); i++)
		k[i] = LE32(((unsigned __int32*)userKey)[i]);

	if (keylen < 32)
		k[keylen/4] |= (unsigned __int32)1 << ((keylen%4)*8);

	k += 8;
	t = k[-1];
	for (i = 0; i < 132; ++i)
		k[i] = t = rotlFixed(k[i-8] ^ k[i-5] ^ k[i-3] ^ t ^ 0x9e3779b9 ^ i, 11);
	k -= 20;

	for (i=0; i<4; i++)
	{
		LKf (k, 20, &a, &e, &b, &d); S3f (&a, &e, &b, &d, &c); SKf (k, 16, &e, &b, &d, &c);
		LKf (k, 24, &c, &b, &a, &e); S2f (&c, &b, &a, &e, &d); SKf (k, 20, &a, &e, &b, &d);
		LKf (k, 28, &b, &e, &c, &a); S1f (&b, &e, &c, &a, &d); SKf (k, 24, &c, &b, &a, &e);
		LKf (k, 32, &a, &b, &c, &d); S0f (&a, &b, &c, &d, &e); SKf (k, 28, &b, &e, &c, &a);
		k += 8*4;
		LKf (k,  4, &a, &c, &d, &b); S7f (&a, &c, &d, &b, &e); SKf (k,  0, &d, &e, &b, &a);
		LKf (k,  8, &a, &c, &b, &e); S6f (&a, &c, &b, &e, &d); SKf (k,  4, &a, &c, &d, &b);
		LKf (k, 12, &b, &a, &e, &c); S5f (&b, &a, &e, &c, &d); SKf (k,  8, &a, &c, &b, &e);
		LKf (k, 16, &e, &b, &d, &c); S4f (&e, &b, &d, &c, &a); SKf (k, 12, &b, &a, &e, &c);
	}
	LKf (k, 20, &a, &e, &b, &d); S3f (&a, &e, &b, &d, &c); SKf (k, 16, &e, &b, &d, &c);
}

#endif // TC_MINIMIZE_CODE_SIZE


#ifndef TC_MINIMIZE_CODE_SIZE

void serpent_encrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	unsigned int i=1;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 8;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	do
	{
		beforeS0(KX); beforeS0(S0); afterS0(LT);
		afterS0(KX); afterS0(S1); afterS1(LT);
		afterS1(KX); afterS1(S2); afterS2(LT);
		afterS2(KX); afterS2(S3); afterS3(LT);
		afterS3(KX); afterS3(S4); afterS4(LT);
		afterS4(KX); afterS4(S5); afterS5(LT);
		afterS5(KX); afterS5(S6); afterS6(LT);
		afterS6(KX); afterS6(S7);

		if (i == 4)
			break;

		++i;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		k += 32;
		beforeS0(LT);
	}
	while (1);

	afterS7(KX);
	
    out[0] = LE32(d);
	out[1] = LE32(e);
	out[2] = LE32(b);
	out[3] = LE32(a);
}

#else // TC_MINIMIZE_CODE_SIZE

typedef unsigned __int32 uint32;

static void LTf (uint32 *a, uint32 *b, uint32 *c, uint32 *d)
{
	*a = rotlFixed(*a, 13);
	*c = rotlFixed(*c, 3);
	*d = rotlFixed(*d ^ *c ^ (*a << 3), 7);
	*b = rotlFixed(*b ^ *a ^ *c, 1);
	*a = rotlFixed(*a ^ *b ^ *d, 5);
	*c = rotlFixed(*c ^ *d ^ (*b << 7), 22);
}

void serpent_encrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	unsigned int i=1;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 8;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	do
	{
		KXf (k,  0, &a, &b, &c, &d); S0f (&a, &b, &c, &d, &e); LTf (&b, &e, &c, &a);
		KXf (k,  4, &b, &e, &c, &a); S1f (&b, &e, &c, &a, &d); LTf (&c, &b, &a, &e);
		KXf (k,  8, &c, &b, &a, &e); S2f (&c, &b, &a, &e, &d); LTf (&a, &e, &b, &d);
		KXf (k, 12, &a, &e, &b, &d); S3f (&a, &e, &b, &d, &c); LTf (&e, &b, &d, &c);
		KXf (k, 16, &e, &b, &d, &c); S4f (&e, &b, &d, &c, &a); LTf (&b, &a, &e, &c);
		KXf (k, 20, &b, &a, &e, &c); S5f (&b, &a, &e, &c, &d); LTf (&a, &c, &b, &e);
		KXf (k, 24, &a, &c, &b, &e); S6f (&a, &c, &b, &e, &d); LTf (&a, &c, &d, &b);
		KXf (k, 28, &a, &c, &d, &b); S7f (&a, &c, &d, &b, &e);

		if (i == 4)
			break;

		++i;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		k += 32;
		LTf (&a,&b,&c,&d);
	}
	while (1);

	KXf (k, 32, &d, &e, &b, &a);
	
    out[0] = LE32(d);
	out[1] = LE32(e);
	out[2] = LE32(b);
	out[3] = LE32(a);
}

#endif // TC_MINIMIZE_CODE_SIZE

#if !defined (TC_MINIMIZE_CODE_SIZE) || defined (TC_WINDOWS_BOOT_SERPENT)

void serpent_decrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 104;
	unsigned int i=4;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	beforeI7(KX);
	goto start;

	do
	{
		c = b;
		b = d;
		d = e;
		k -= 32;
		beforeI7(ILT);
start:
		beforeI7(I7); afterI7(KX); 
		afterI7(ILT); afterI7(I6); afterI6(KX); 
		afterI6(ILT); afterI6(I5); afterI5(KX); 
		afterI5(ILT); afterI5(I4); afterI4(KX); 
		afterI4(ILT); afterI4(I3); afterI3(KX); 
		afterI3(ILT); afterI3(I2); afterI2(KX); 
		afterI2(ILT); afterI2(I1); afterI1(KX); 
		afterI1(ILT); afterI1(I0); afterI0(KX);
	}
	while (--i != 0);
	
    out[0] = LE32(a);
	out[1] = LE32(d);
	out[2] = LE32(b);
	out[3] = LE32(e);
}

#else // TC_MINIMIZE_CODE_SIZE && !TC_WINDOWS_BOOT_SERPENT

static void ILTf (uint32 *a, uint32 *b, uint32 *c, uint32 *d)
{ 
	*c = rotrFixed(*c, 22);
	*a = rotrFixed(*a, 5);
	*c ^= *d ^ (*b << 7);
	*a ^= *b ^ *d;
	*b = rotrFixed(*b, 1);
	*d = rotrFixed(*d, 7) ^ *c ^ (*a << 3);
	*b ^= *a ^ *c;
	*c = rotrFixed(*c, 3);
	*a = rotrFixed(*a, 13);
}

void serpent_decrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 104;
	unsigned int i=4;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	KXf (k, 32, &a, &b, &c, &d);
	goto start;

	do
	{
		c = b;
		b = d;
		d = e;
		k -= 32;
		beforeI7(ILT);
start:
		beforeI7(I7); KXf (k, 28, &d, &a, &b, &e);
		ILTf (&d, &a, &b, &e); afterI7(I6); KXf (k, 24, &a, &b, &c, &e); 
		ILTf (&a, &b, &c, &e); afterI6(I5); KXf (k, 20, &b, &d, &e, &c); 
		ILTf (&b, &d, &e, &c); afterI5(I4); KXf (k, 16, &b, &c, &e, &a); 
		ILTf (&b, &c, &e, &a); afterI4(I3); KXf (k, 12, &a, &b, &e, &c);
		ILTf (&a, &b, &e, &c); afterI3(I2); KXf (k, 8,  &b, &d, &e, &c);
		ILTf (&b, &d, &e, &c); afterI2(I1); KXf (k, 4,  &a, &b, &c, &e);
		ILTf (&a, &b, &c, &e); afterI1(I0); KXf (k, 0,  &a, &d, &b, &e);
	}
	while (--i != 0);
	
    out[0] = LE32(a);
	out[1] = LE32(d);
	out[2] = LE32(b);
	out[3] = LE32(e);
}

#endif // TC_MINIMIZE_CODE_SIZE && !TC_WINDOWS_BOOT_SERPENT
